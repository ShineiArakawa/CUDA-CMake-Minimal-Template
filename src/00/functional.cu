#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <hip/hip_runtime.h>
#include <>

#include <00/functional.cuh>
#include <cassert>
#include <chrono>
#include <cmath>
#include <iostream>

namespace utils {
int divRoundUp(int value, int radix) { return (value + radix - 1) / radix; };

void deviceInfo(int deviceID) {
  hipDeviceProp_t dProp;
  hipGetDeviceProperties(&dProp, deviceID);
  std::cout << "########################## CUDA Device Info "
               "##########################"
            << std::endl;
  std::cout << "Device: " << dProp.name << std::endl;
  std::cout << "Maximum number of threads per block:"
            << dProp.maxThreadsPerBlock << std::endl;
  std::cout << "Max dimension size of a thread block (x,y,z): ("
            << dProp.maxThreadsDim[0] << "," << dProp.maxThreadsDim[1] << ","
            << dProp.maxThreadsDim[2] << ")" << std::endl;
  std::cout << "Max dimension size of a grid size    (x,y,z): ("
            << dProp.maxGridSize[0] << "," << dProp.maxGridSize[1] << ","
            << dProp.maxGridSize[2] << ")" << std::endl;
  std::cout << "###############################################################"
               "#######"
            << std::endl;
}

Matrix::Matrix() {
  a = 1;
  b = 0;
  c = 0;
  d = 1;
  tx = 0;
  ty = 0;
}

Matrix::~Matrix() {}

Matrix Matrix::getRotMat(const float degree) {
  Matrix rotMat;

  const float rad = degree / 180.0f * M_PI;

  rotMat.a = std::cos(rad);
  rotMat.b = -std::sin(rad);
  rotMat.c = std::sin(rad);
  rotMat.d = std::cos(rad);

  return rotMat;
}

Matrix Matrix::inv() {
  Matrix inverted;

  float det = a * d - b * c;

  assert(det > 0.0f);

  inverted.a = d / det;
  inverted.b = -b / det;
  inverted.c = -c / det;
  inverted.d = a / det;
  inverted.tx = (b * ty - d * tx) / det;
  inverted.ty = (c * tx - a * ty) / det;

  return inverted;
}

std::string Matrix::to_string() {
  std::string buffer = "";
  buffer += std::to_string(a) + " " + std::to_string(b) + " " +
            std::to_string(tx) + "\n";
  buffer += std::to_string(c) + " " + std::to_string(d) + " " +
            std::to_string(ty) + "\n";
  buffer += std::to_string(0) + " " + std::to_string(0) + " " +
            std::to_string(1) + "\n";
  return buffer;
}
}  // namespace utils

namespace cKernel {
__global__ void convertToGrayKernel(uchar3 *inPixel,
                                    unsigned char *outinPixel) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  outinPixel[index] = (unsigned char)(0.299f * inPixel[index].x +
                                      0.587f * (float)inPixel[index].y +
                                      0.114f * (float)inPixel[index].z);
}

__global__ void invertKernel(uchar3 *inPixel, uchar3 *outinPixel) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  outinPixel[index].x = 255 - inPixel[index].x;
  outinPixel[index].y = 255 - inPixel[index].y;
  outinPixel[index].z = 255 - inPixel[index].z;
}

__global__ void warpKernel(const utils::Matrix transMat, const uchar4 *input,
                           uchar4 *output, const int width, const int height,
                           const int pitch) {
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  if ((x < width) && (y < height)) {
    const float xOffset = 0.5f - width * 0.5f;
    const float yOffset = 0.5f - height * 0.5f;

    const float xOut = x + xOffset;
    const float yOut = y + yOffset;

    const float xIn =
        transMat.a * xOut + transMat.b * yOut + transMat.tx - xOffset;
    const float yIn =
        transMat.c * xOut + transMat.d * yOut + transMat.tx - yOffset;

    uchar4 value;
    if ((0.0f <= xIn) && (xIn < width) && (0.0f <= yIn) && (yIn < height)) {
      const int inIndex = (int)xIn + pitch * (int)yIn;
      value = input[inIndex];
    } else {
      value = make_uchar4(0, 0, 0, 0);
    }

    const int outIndex = x + pitch * y;
    output[outIndex] = value;
  }
}

}  // namespace cKernel

namespace cLaunch {
common::Image cudaWarpTransform(common::Image image, utils::Matrix &transMat) {
  const int height = image.getHeight();
  const int width = image.getWidth();
  const int channels = image.getChannles();
  assert(channels == 3);

  // Data format convertion
  uchar4 *hostImageIn = new uchar4[width * height];
  uchar4 *hostImageOut = new uchar4[width * height];

  for (int h = 0; h < height; h++) {
    for (int w = 0; w < width; w++) {
      hostImageIn[w + h * width].x = image.getPixel(w, h, 0);
      hostImageIn[w + h * width].y = image.getPixel(w, h, 1);
      hostImageIn[w + h * width].z = image.getPixel(w, h, 2);
      hostImageIn[w + h * width].w = 255;
    }
  }

  uchar4 *deviceImageIn;
  uchar4 *deviceImageOut;
  utils::Matrix invTransMat = transMat.inv();

  {
    // Allocate device memory
    size_t hostPitch = sizeof(uchar4) * width;
    size_t devicePitch;
    hipMallocPitch(&deviceImageIn, &devicePitch, sizeof(uchar4) * width,
                    height);
    hipMallocPitch(&deviceImageOut, &devicePitch, sizeof(uchar4) * width,
                    height);
    const size_t pitchesInPixel = devicePitch / sizeof(uchar4);

    // Transfer (CPU → GPU)
    hipMemcpy2D(deviceImageIn, devicePitch, hostImageIn, hostPitch, hostPitch,
                 height, hipMemcpyHostToDevice);

    std::vector<int> nBlocksPower = {6, 7, 8, 9, 10};
    std::vector<std::vector<int>> blockPiars;
    for (int power : nBlocksPower) {
      for (int i = 0; i < power + 1; i++) {
        blockPiars.push_back(
            {(int)std::pow(2, i), (int)std::pow(2, power - i)});
      }
    }

    for (auto blockPiar : blockPiars) {
      dim3 blockDim(blockPiar[0], blockPiar[1]);
      dim3 gridDim(utils::divRoundUp(width, blockDim.x),
                   utils::divRoundUp(height, blockDim.y));

      // Launch
      std::chrono::system_clock::time_point start, end;
      start = std::chrono::system_clock::now();
      {
        int nTimes = std::pow(2, 17);
        for (int i = 0; i < nTimes; i++) {
          cKernel::warpKernel<<<gridDim, blockDim>>>(invTransMat, deviceImageIn,
                                                     deviceImageOut, width,
                                                     height, pitchesInPixel);
        }
      }
      end = std::chrono::system_clock::now();
      double elapsed =
          std::chrono::duration_cast<std::chrono::milliseconds>(end - start)
              .count();
      std::cout << "blockDim: (" << std::to_string(blockDim.x) << ", "
                << std::to_string(blockDim.y) << "), Kernel Execution Time: "
                << std::to_string(elapsed / 1000.0) << " [sec]" << std::endl;
    }

    // Transfer (GPU → CPU)
    hipMemcpy2D(hostImageOut, hostPitch, deviceImageOut, devicePitch,
                 hostPitch, height, hipMemcpyDeviceToHost);

    // Error check
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != 0) {
      std::cout << "Raised CUDA Error !" << std::endl;
    }
  }

  common::Image outImage = common::Image();
  outImage.newImage(width, height, channels);

  for (int h = 0; h < height; h++) {
    for (int w = 0; w < width; w++) {
      outImage.setPixel(w, h, 0, hostImageOut[w + h * width].x);
      outImage.setPixel(w, h, 1, hostImageOut[w + h * width].y);
      outImage.setPixel(w, h, 2, hostImageOut[w + h * width].z);
    }
  }

  hipFree(deviceImageIn);
  hipFree(deviceImageOut);
  free(hostImageIn);
  free(hostImageOut);

  return outImage;
}

common::Image cudaGrayScaleTransform(common::Image image) {
  int height = image.getHeight();
  int width = image.getWidth();
  int channels = image.getChannles();

  assert(channels == 3);

  uchar3 *hostImageRGB = new uchar3[width * height];
  unsigned char *hostImageGray = new unsigned char[width * height];

  for (int h = 0; h < height; h++) {
    for (int w = 0; w < width; w++) {
      hostImageRGB[w + h * width].x = image.getPixel(w, h, 0);
      hostImageRGB[w + h * width].y = image.getPixel(w, h, 1);
      hostImageRGB[w + h * width].z = image.getPixel(w, h, 2);
    }
  }

  uchar3 *deviceImageRGB;
  unsigned char *deviceImageGray;
  int dataSizeRGB = sizeof(uchar3) * width * height;
  int dataSizeGray = sizeof(unsigned char) * width * height;
  hipMalloc((void **)&deviceImageRGB, dataSizeRGB);
  hipMalloc((void **)&deviceImageGray, dataSizeGray);

  // Transfer (CPU → GPU)
  hipMemcpy(deviceImageRGB, hostImageRGB, dataSizeRGB, hipMemcpyHostToDevice);

  cKernel::convertToGrayKernel<<<width * height, 1>>>(deviceImageRGB,
                                                      deviceImageGray);

  // Transfer(GPU → CPU)
  hipMemcpy(hostImageGray, deviceImageGray, dataSizeGray,
             hipMemcpyDeviceToHost);

  common::Image grayImage = common::Image();
  grayImage.newImage(width, height, channels);

  for (int h = 0; h < height; h++) {
    for (int w = 0; w < width; w++) {
      grayImage.setPixel(w, h, 0, hostImageGray[w + h * width]);
      grayImage.setPixel(w, h, 1, hostImageGray[w + h * width]);
      grayImage.setPixel(w, h, 2, hostImageGray[w + h * width]);
    }
  }

  hipFree(deviceImageRGB);
  hipFree(deviceImageGray);
  free(hostImageRGB);
  free(hostImageGray);

  return grayImage;
}

common::Image cudaInvertTransform(common::Image image) {
  int height = image.getHeight();
  int width = image.getWidth();
  int channels = image.getChannles();

  assert(channels == 3);

  uchar3 *hostImageIn = new uchar3[width * height];
  uchar3 *hostImageOut = new uchar3[width * height];

  for (int h = 0; h < height; h++) {
    for (int w = 0; w < width; w++) {
      hostImageIn[w + h * width].x = image.getPixel(w, h, 0);
      hostImageIn[w + h * width].y = image.getPixel(w, h, 1);
      hostImageIn[w + h * width].z = image.getPixel(w, h, 2);
    }
  }

  uchar3 *deviceImageIn;
  uchar3 *deviceImageOut;
  int dataSizeIn = sizeof(uchar3) * width * height;
  int dataSizeOut = sizeof(uchar3) * width * height;
  hipMalloc((void **)&deviceImageIn, dataSizeIn);
  hipMalloc((void **)&deviceImageOut, dataSizeOut);

  // Transfer (CPU → GPU)
  hipMemcpy(deviceImageIn, hostImageIn, dataSizeIn, hipMemcpyHostToDevice);

  cKernel::invertKernel<<<width * height, 1>>>(deviceImageIn, deviceImageOut);

  // Transfer (GPU → CPU)
  hipMemcpy(hostImageOut, deviceImageOut, dataSizeOut, hipMemcpyDeviceToHost);

  common::Image outImage = common::Image();
  outImage.newImage(width, height, channels);

  for (int h = 0; h < height; h++) {
    for (int w = 0; w < width; w++) {
      outImage.setPixel(w, h, 0, hostImageOut[w + h * width].x);
      outImage.setPixel(w, h, 1, hostImageOut[w + h * width].y);
      outImage.setPixel(w, h, 2, hostImageOut[w + h * width].z);
    }
  }

  hipFree(deviceImageIn);
  hipFree(deviceImageOut);
  free(hostImageIn);
  free(hostImageOut);

  hipError_t hipError_t = hipGetLastError();
  std::cout << "hipError_t=" << hipError_t << std::endl;

  return outImage;
}
}  // namespace cLaunch